#include "hip/hip_runtime.h"
#include <cstddef>
#include <cstdio>
#if _XOPEN_SOURCE < 600
#include <__clang_cuda_builtin_vars.h>
#define _XOPEN_SOURCE 600
#endif

#include <assert.h>
#include <stdio.h>
#include <stdlib.h>

#include "hpc.h"

#define BLOCKDIM 1024

typedef struct
{
    float *P; /* coordinates P[i][j] of point i               */
    int N;    /* Number of points (rows of matrix P)          */
    int D;    /* Number of dimensions (columns of matrix P)   */
} points_t;

/**
 * Read input from stdin. Input format is:
 *
 * d [other ignored stuff]
 * N
 * p0,0 p0,1 ... p0,d-1
 * p1,0 p1,1 ... p1,d-1
 * ...
 * pn-1,0 pn-1,1 ... pn-1,d-1
 *
 */
void read_input(points_t *points)
{
    char buf[1024];
    int N, D;
    float *P;

    if (1 != scanf("%d", &D))
    {
        fprintf(stderr, "FATAL: can not read the dimension\n");
        exit(EXIT_FAILURE);
    }
    assert(D >= 2);
    if (NULL == fgets(buf, sizeof(buf), stdin))
    { /* ignore rest of the line */
        fprintf(stderr, "FATAL: can not read the first line\n");
        exit(EXIT_FAILURE);
    }
    if (1 != scanf("%d", &N))
    {
        fprintf(stderr, "FATAL: can not read the number of points\n");
        exit(EXIT_FAILURE);
    }
    P = (float *)malloc(D * N * sizeof(*P));
    assert(P);
    for (int i = 0; i < N; i++)
    {
        for (int k = 0; k < D; k++)
        {
            if (1 != scanf("%f", &(P[i * D + k])))
            {
                fprintf(stderr, "FATAL: failed to get coordinate %d of point %d\n", k, i);
                exit(EXIT_FAILURE);
            }
        }
    }
    points->P = P;
    points->N = N;
    points->D = D;
}

void free_points(points_t *points)
{
    free(points->P);
    points->P = NULL;
    points->N = points->D = -1;
}

/* Returns 1 if |p| dominates |q| */
__device__ int dominates(const float *p, const float *q, int D)
{
    /* The following loops could be merged, but the keep them separated
       for the sake of readability */
    for (int k = 0; k < D; k++)
    {
        if (p[k] < q[k])
        {
            return 0;
        }
    }
    for (int k = 0; k < D; k++)
    {
        if (p[k] > q[k])
        {
            return 1;
        }
    }
    return 0;
}

/**
 * Print the coordinates of points belonging to the skyline `s` to
 * standard ouptut. `s[i] == 1` if point `i` belongs to the skyline.
 * The output format is the same as the input format, so that this
 * program can process its own output.
 */
void print_skyline(const points_t *points, const int *s, int r)
{
    const int D = points->D;
    const int N = points->N;
    const float *P = points->P;

    printf("%d\n", D);
    printf("%d\n", r);
    for (int i = 0; i < N; i++)
    {
        if (s[i])
        {
            for (int k = 0; k < D; k++)
            {
                printf("%f ", P[i * D + k]);
            }
            printf("\n");
        }
    }
}

__constant__ int d_N;

__global__ void ker_init(int *s)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < d_N)
    {
        s[index] = 1;
    }
}

/*
__device__ int d_i = 0;

__global__ void ker_skyline_all(float *p, int *s)
{
    const int bindex = blockIdx.x;
    const int tindex = threadIdx.x;

    int elem = tindex + bindex * BLOCKDIM;

    if (elem >= d_N)
    {
        return;
    }

    while (d_i < d_N)
    {
        if (s[d_i])
        {
            if (dominates(&(p[d_i * d_D]), &(p[elem * d_D]), d_D))
            {
                s[elem] = 0;
                atomicAdd(&d_its, 1);
            }
        }
        __syncthreads();
        if (elem == 0)
        {
            d_i++;
        }
        __syncthreads();
    }

    if (elem == 0)
    {
        printf("Its: %d\n", d_i);
    }
}
*/

__constant__ int d_D;
__constant__ int d_r;
__device__ int d_its;

__global__ void ker_single_skyline(float *p, int *s)
{
    const int bindex = blockIdx.x;
    const int tindex = threadIdx.x;

    int elem = tindex + bindex * BLOCKDIM;

    if (elem >= d_N)
    {
        return;
    }

    for (int i = 0; i < d_N; i++)
    {
        if (s[i])
        {
            if (s[elem] && dominates(&(p[i * d_D]), &(p[elem * d_D]), d_D))
            {
                s[elem] = 0;
            }
        }
        __syncthreads();
    }
}

int main(int argc, char *argv[])
{
    points_t points;
    int its = 0;

    float *d_points;
    int *d_s;

    if (argc != 1)
    {
        fprintf(stderr, "Usage: %s < input_file > output_file\n", argv[0]);
        return EXIT_FAILURE;
    }

    read_input(&points);
    int *s = (int *)malloc(points.N * sizeof(*s));
    assert(s);

    const size_t size_points = points.D * points.N * sizeof(float);
    hipMalloc((void **)&d_points, size_points);
    fprintf(stderr, "Points memory allocated: %zu\n", size_points);

    const size_t size_s = points.N * sizeof(int);
    hipMalloc((void **)&d_s, size_s);
    fprintf(stderr, "s array memory allocated: %zu\n", size_s);

    // copy points to GPU memory
    hipMemcpy(d_points, points.P, size_points, hipMemcpyHostToDevice);
    fprintf(stderr, "Points copied\n");

    // declare global variables
    hipMemcpyToSymbol(HIP_SYMBOL(d_N), &points.N, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(d_D), &points.D, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(d_r), &points.N, sizeof(int));

    // init s array
    const double istart = hpc_gettime();
    ker_init<<<(points.N + BLOCKDIM - 1) / BLOCKDIM, BLOCKDIM>>>(d_s);
    const double ielasped = hpc_gettime() - istart;
    fprintf(stderr, "Init time: %f\n", ielasped);

    const double tstart = hpc_gettime();
    const int blocks = (points.N + BLOCKDIM - 1) / BLOCKDIM;
    fprintf(stderr, "%d blocks\n", blocks);
    // ker_skyline_all<<<blocks, BLOCKDIM>>>(d_points, d_s);

    ker_single_skyline<<<blocks, BLOCKDIM>>>(d_points, d_s);

    hipMemcpy(s, d_s, size_s, hipMemcpyDeviceToHost);

    int r = 0;
    for (int i = 0; i < points.N; i++)
    {
        r += s[i];
    }

    const double elapsed = hpc_gettime() - tstart;
    hipMemcpyFromSymbol(&its, HIP_SYMBOL(d_its), sizeof(int));

    // print_skyline(&points, s, r);

    fprintf(stderr, "\n\t%d points\n", points.N);
    fprintf(stderr, "\t%d dimensions\n", points.D);
    fprintf(stderr, "\t%d points in skyline\n", r);
    fprintf(stderr, "\t%d iterations\n\n", its);
    fprintf(stderr, "Execution time (s) %f\n", elapsed);
    printf("%f", elapsed);

    hipFree(d_points);
    hipFree(d_s);

    free_points(&points);
    free(s);
    return EXIT_SUCCESS;
}
