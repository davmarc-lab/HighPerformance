#include "hip/hip_runtime.h"
#include <cstddef>
#include <cstdio>
#if _XOPEN_SOURCE < 600
#include <__clang_cuda_builtin_vars.h>
#define _XOPEN_SOURCE 600
#endif

#include <assert.h>
#include <stdio.h>
#include <stdlib.h>

#include "hpc.h"

#define BLOCKDIM 1024

typedef struct
{
    float *P; /* coordinates P[i][j] of point i               */
    int N;    /* Number of points (rows of matrix P)          */
    int D;    /* Number of dimensions (columns of matrix P)   */
} points_t;

/**
 * Read input from stdin. Input format is:
 *
 * d [other ignored stuff]
 * N
 * p0,0 p0,1 ... p0,d-1
 * p1,0 p1,1 ... p1,d-1
 * ...
 * pn-1,0 pn-1,1 ... pn-1,d-1
 *
 */
void read_input(points_t *points)
{
    char buf[1024];
    int N, D;
    float *P;

    if (1 != scanf("%d", &D))
    {
        fprintf(stderr, "FATAL: can not read the dimension\n");
        exit(EXIT_FAILURE);
    }
    assert(D >= 2);
    if (NULL == fgets(buf, sizeof(buf), stdin))
    { /* ignore rest of the line */
        fprintf(stderr, "FATAL: can not read the first line\n");
        exit(EXIT_FAILURE);
    }
    if (1 != scanf("%d", &N))
    {
        fprintf(stderr, "FATAL: can not read the number of points\n");
        exit(EXIT_FAILURE);
    }
    P = (float *)malloc(D * N * sizeof(*P));
    assert(P);
    for (int i = 0; i < N; i++)
    {
        for (int k = 0; k < D; k++)
        {
            if (1 != scanf("%f", &(P[i * D + k])))
            {
                fprintf(stderr, "FATAL: failed to get coordinate %d of point %d\n", k, i);
                exit(EXIT_FAILURE);
            }
        }
    }
    points->P = P;
    points->N = N;
    points->D = D;
}

void free_points(points_t *points)
{
    free(points->P);
    points->P = NULL;
    points->N = points->D = -1;
}

/* Returns 1 if |p| dominates |q| */
__device__ int dominates(const float *p, const float *q, int D)
{
    /* The following loops could be merged, but the keep them separated
       for the sake of readability */
    for (int k = 0; k < D; k++)
    {
        if (p[k] < q[k])
        {
            return 0;
        }
    }
    for (int k = 0; k < D; k++)
    {
        if (p[k] > q[k])
        {
            return 1;
        }
    }
    return 0;
}

/**
 * Print the coordinates of points belonging to the skyline `s` to
 * standard ouptut. `s[i] == 1` if point `i` belongs to the skyline.
 * The output format is the same as the input format, so that this
 * program can process its own output.
 */
void print_skyline(const points_t *points, const int *s, int r)
{
    const int D = points->D;
    const int N = points->N;
    const float *P = points->P;

    printf("%d\n", D);
    printf("%d\n", r);
    for (int i = 0; i < N; i++)
    {
        if (s[i])
        {
            for (int k = 0; k < D; k++)
            {
                printf("%f ", P[i * D + k]);
            }
            printf("\n");
        }
    }
}

__constant__ int d_N;

__global__ void ker_init(int *s)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < d_N)
    {
        s[index] = 1;
    }
}

__constant__ int d_D;
__constant__ int d_r;
__device__ int d_its;

__global__ void ker_single_skyline(float *p, int *s)
{
    const int bindex = blockIdx.x;
    const int tindex = threadIdx.x;

    int elem = tindex + bindex * BLOCKDIM;

    if (elem >= d_N)
    {
        return;
    }

    for (int i = 0; i < d_N; i++)
    {
        if (s[i])
        {
            if (s[elem] && dominates(&(p[i * d_D]), &(p[elem * d_D]), d_D))
            {
                s[elem] = 0;
                atomicAdd(&d_its, 1);
            }
        }
        __syncthreads();
    }
}

int main(int argc, char *argv[])
{
    points_t points;
    int its = 0;

    float *d_points;
    int *d_s;

    if (argc != 1)
    {
        fprintf(stderr, "Usage: %s < input_file > output_file\n", argv[0]);
        return EXIT_FAILURE;
    }

    read_input(&points);
    int *s = (int *)malloc(points.N * sizeof(*s));
    assert(s);

    const size_t size_points = points.D * points.N * sizeof(float);
    const size_t size_s = points.N * sizeof(int);

    fprintf(stderr, "Allocating GPU memory\n");
    const double astart = hpc_gettime();

    hipMalloc((void **)&d_points, size_points);
    fprintf(stderr, "\t'points' memory allocated: %zu\n", size_points);

    hipMalloc((void **)&d_s, size_s);
    fprintf(stderr, "\t's' array memory allocated: %zu\n", size_s);
    const double aelapsed = hpc_gettime() - astart;
    fprintf(stderr, "\tMalloc time: %lf s\n\n", aelapsed);

    // copy points to GPU memory
    fprintf(stderr, "Copying data\n");
    const double cstart = hpc_gettime();

    hipMemcpy(d_points, points.P, size_points, hipMemcpyHostToDevice);
    fprintf(stderr, "\t'points' copied\n");

    // declare global variables
    hipMemcpyToSymbol(HIP_SYMBOL(d_N), &points.N, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(d_D), &points.D, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(d_r), &points.N, sizeof(int));
    const double celapsed = hpc_gettime() - cstart;
    fprintf(stderr, "\tCopy time: %lf s\n\n", celapsed);

    // init s array
    const double istart = hpc_gettime();
    ker_init<<<(points.N + BLOCKDIM - 1) / BLOCKDIM, BLOCKDIM>>>(d_s);
    const double ielasped = hpc_gettime() - istart;
    fprintf(stderr, "'s' init time: %lf s\n\n", ielasped);

    const int blocks = (points.N + BLOCKDIM - 1) / BLOCKDIM;
    fprintf(stderr, "%d blocks, %d thread per block\n", blocks, BLOCKDIM);

    fprintf(stderr, "\nStart skyline:\n");

    const double tstart = hpc_gettime();
    ker_single_skyline<<<blocks, BLOCKDIM>>>(d_points, d_s);
    hipDeviceSynchronize();
    fprintf(stderr, "-- GPU call finished t => %lf s\n", hpc_gettime() - tstart);
    fprintf(stderr, "-- cpoying s and start final reduction iteration\n");

    hipMemcpy(s, d_s, size_s, hipMemcpyDeviceToHost);

    int r = 0;
    for (int i = 0; i < points.N; i++)
    {
        r += s[i];
    }

    const double elapsed = hpc_gettime() - tstart;
    hipMemcpyFromSymbol(&its, HIP_SYMBOL(d_its), sizeof(int));

    // print_skyline(&points, s, r);

    fprintf(stderr, "\n\t%d points\n", points.N);
    fprintf(stderr, "\t%d dimensions\n", points.D);
    fprintf(stderr, "\t%d points in skyline\n", r);
    fprintf(stderr, "\t%d iterations\n\n", its);
    fprintf(stderr, "Execution time (s) %f\n", elapsed);
    printf("%f", elapsed);

    hipFree(d_points);
    hipFree(d_s);

    free_points(&points);
    free(s);
    return EXIT_SUCCESS;
}
